
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello()
{
    printf("Hello World from thread %d %d\n", threadIdx.x, threadIdx.y);
}

int main()
{
    dim3 threadsPerBlock(3, 3);
    cuda_hello<<<1, threadsPerBlock>>>();
    hipDeviceSynchronize();
    return 0;
}
