
#include <hip/hip_runtime.h>
#include <stdio.h>

// Definición de una función de kernel CUDA
__global__ void cuda_hello()
{
    // Impresión de un mensaje desde el kernel
    printf("Hello World from GPU!\n");
}

// Función principal
int main()
{
    // Llamada al kernel con una configuración de 1 bloque y 1 hilo
    cuda_hello<<<1, 1>>>();

    // Sincronización del dispositivo para asegurarse de que todos los kernels hayan terminado
    hipDeviceSynchronize();

    return 0;
}